#include "hip/hip_runtime.h"
/* =======================================================================================
   This file is released as part of GraviT - scalable, platform independent ray
   tracing
   tacc.github.io/GraviT

   Copyright 2013-2015 Texas Advanced Computing Center, The University of Texas
   at Austin
   All rights reserved.

   Licensed under the BSD 3-Clause License, (the "License"); you may not use
   this file
   except in compliance with the License.
   A copy of the License is included with this software in the file LICENSE.
   If your copy does not contain the License, you may obtain a copy of the
   License at:

       http://opensource.org/licenses/BSD-3-Clause

   Unless required by applicable law or agreed to in writing, software
   distributed under
   the License is distributed on an "AS IS" BASIS, WITHOUT WARRANTIES OR
   CONDITIONS OF ANY
   KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under
   limitations under the License.

   GraviT is funded in part by the US National Science Foundation under awards
   ACI-1339863,
   ACI-1339881 and ACI-1339840
   =======================================================================================
   */
//
// OptixMeshAdapter.cu
//

#include <float.h>
#include "Mesh.cuh"
#include "Ray.cuh"
#include "Light.cuh"
#include "Material.cuh"
#include "hip/hip_runtime.h"
#include "OptixMeshAdapter.cuh"
#include "cutil_math.h"

__device__ hiprandState *globalState;

using namespace gvt;
using namespace render;
using namespace data;
using namespace cuda_primitives;

__device__ int getGlobalIdx_2D_2D() {
  int blockId = blockIdx.x + blockIdx.y * gridDim.x;
  int threadId = blockId * (blockDim.x * blockDim.y) +
                 (threadIdx.y * blockDim.x) + threadIdx.x;
  return threadId;

}

__global__ void setup_kernel(hiprandState *state, unsigned long seed) {
  int id = getGlobalIdx_2D_2D();
  hiprand_init(seed, id, 0, &state[id]);

  if (id == 0)
    globalState = state;
}

__device__ float cudaRand() {

  float RANDOM;

  int ind = getGlobalIdx_2D_2D();
  hiprandState localState = globalState[ind];
  RANDOM = hiprand_uniform(&localState);
  globalState[ind] = localState;

  return RANDOM;
}

hiprandState *set_random_states(int rayCount) {


	 dim3 threadsPerBlock = dim3(16, 16);
	  dim3 numBlocks = dim3((rayCount / (threadsPerBlock.x * threadsPerBlock.y)) + 1, 1);

  int N = numBlocks.x * numBlocks.y * threadsPerBlock.x * threadsPerBlock.y;
  hiprandState *devStates;
  hipMalloc(&devStates, N * sizeof(hiprandState));

  // setup seeds
  setup_kernel<<<numBlocks, threadsPerBlock>>>(devStates, time(NULL));
  gpuErrchk(hipGetLastError());

  return devStates;
}

__global__ void cudaKernelPrepOptixRays(OptixRay* optixrays, bool* valid,
                   const int localPacketSize,
                   Ray* rays,
                   CudaGvtContext* cudaGvtCtx, bool ignoreValid) {

	int i = getGlobalIdx_2D_2D();
	if (i >= localPacketSize) return;

    if (ignoreValid || valid[i]) {
       Ray &r = rays[ i];

      r.origin.w=1;
      float4 origin = (*(cudaGvtCtx->minv)) * r.origin; // transform ray to local space
      float4 direction = (*(cudaGvtCtx->minv)) * r.direction;

      OptixRay optix_ray;
      optix_ray.origin[0] = origin.x;
      optix_ray.origin[1] = origin.y;
      optix_ray.origin[2] = origin.z;
      optix_ray.t_min = 0;
      optix_ray.direction[0] = direction.x;
      optix_ray.direction[1] = direction.y;
      optix_ray.direction[2] = direction.z;
      optix_ray.t_max = FLT_MAX;
      optixrays[i] = optix_ray;


    }
}

void cudaPrepOptixRays(OptixRay* optixrays, bool* valid,
                  const int localPacketSize, Ray* rays,
                 CudaGvtContext* cudaGvtCtx,bool ignoreValid, hipStream_t& stream) {

		dim3 blockDIM = dim3(16, 16);
		dim3 gridDIM = dim3((localPacketSize / (blockDIM.x * blockDIM.y)) + 1, 1);


		cudaKernelPrepOptixRays<<<gridDIM,blockDIM , 0, stream>>>(
				optixrays,valid,localPacketSize,rays,
				cudaGvtCtx->toGPU(),ignoreValid);

		  gpuErrchk(hipGetLastError());


}

__global__ void cudaKernelFilterShadow( CudaGvtContext* cudaGvtCtx, const int startingDepth) {

	int tID = getGlobalIdx_2D_2D();
	if (tID >= cudaGvtCtx->shadowRayCount) return;

	 if (cudaGvtCtx->traceHits[tID].triangle_id < 0) {
	          // ray is valid, but did not hit anything, so add to dispatch queue
	    	  int a = atomicAdd((int *)&(cudaGvtCtx->dispatchCount), 1);
	    	  cudaGvtCtx->dispatch[a] = cudaGvtCtx->shadowRays[tID];
	        }
    else if (cudaGvtCtx->shadowRays[tID].depth == startingDepth)
    {
      cudaGvtCtx->shadowRays[tID].type = gvt::render::data::cuda_primitives::Ray::OCCLUDED;
      int a = atomicAdd((int *)&(cudaGvtCtx->dispatchCount), 1);
      cudaGvtCtx->dispatch[a] = cudaGvtCtx->shadowRays[tID];       
    }
}

void cudaProcessShadows(CudaGvtContext* cudaGvtCtx, const int startingDepth) {

		dim3 blockDIM = dim3(16, 16);
		dim3 gridDIM = dim3((cudaGvtCtx->shadowRayCount / (blockDIM.x * blockDIM.y)) + 1, 1);

		cudaKernelFilterShadow<<<gridDIM,blockDIM , 0, cudaGvtCtx->stream>>>(cudaGvtCtx->toGPU(), startingDepth);
		gpuErrchk(hipGetLastError());

		cudaGvtCtx->toHost();
}



__device__ void generateShadowRays(const Ray &r, const float4 &normal,
                                   int primID, CudaGvtContext* cudaGvtCtx) {

  for (int l = 0; l < cudaGvtCtx->nLights; l++) {


    Light *light = &(cudaGvtCtx->lights[l]);

    // Try to ensure that the shadow ray is on the correct side of the
    // triangle.
    // Technique adapted from "Robust BVH Ray Traversal" by Thiago Ize.
    // Using about 8 * ULP(t).
    const float multiplier = 1.0f - 16.0f * FLT_EPSILON;
    const float t_shadow = multiplier * r.t;

    float4 origin = r.origin + r.direction * t_shadow;
    origin.w=1.0f;
    const float4 dir = light->light.position - origin;
    const float t_max = length(dir);

    Ray shadow_ray;

    shadow_ray.origin = origin;
    shadow_ray.setDirection(dir);
    shadow_ray.w = r.w;
    shadow_ray.type = Ray::SHADOW;
    shadow_ray.depth = r.depth;
    shadow_ray.t = r.t;
    shadow_ray.id = r.id;
    shadow_ray.t_max = t_max;

    Color c = cudaGvtCtx->mesh.mat->shade(/*primID,*/ shadow_ray, normal, light);


    shadow_ray.color.x = c.x;
    shadow_ray.color.y = c.y;
    shadow_ray.color.z = c.z;
    shadow_ray.color.w = 1.0f;

    int a = atomicAdd((int *)&(cudaGvtCtx->shadowRayCount), 1);
    cudaGvtCtx->shadowRays[a] = shadow_ray;


  }
}



__global__ void kernel(gvt::render::data::cuda_primitives::CudaGvtContext* cudaGvtCtx,  const int startingDepth) {

	int tID = getGlobalIdx_2D_2D();

	if (tID >= cudaGvtCtx->rayCount) return;

    if (cudaGvtCtx->valid[tID]) {
      Ray &r = cudaGvtCtx->rays[tID];
      if (cudaGvtCtx->traceHits[tID].triangle_id >= 0) {

        // ray has hit something
        // shadow ray hit something, so it should be dropped
        if (r.type == Ray::SHADOW) {
          if(r.depth == startingDepth)
          {
            r.type = gvt::render::data::cuda_primitives::Ray::OCCLUDED;
            int a = atomicAdd((int *)&(cudaGvtCtx->dispatchCount), 1);
            cudaGvtCtx->dispatch[a] = r;
          }
        	return;
        }

        float t = cudaGvtCtx->traceHits[tID].t;
        r.t = t;

        float4 manualNormal;
        {
          const int triangle_id = cudaGvtCtx->traceHits[tID].triangle_id;
#ifndef FLAT_SHADING
          const float u = cudaGvtCtx->traceHits[tID].u;
          const float v = cudaGvtCtx->traceHits[tID].v;
          const int3 &normals =
        		  cudaGvtCtx->mesh.faces_to_normals[triangle_id]; // FIXME: need to
                                                   // figure out
                                                   // to store
          // `faces_to_normals`
          // list
          const float4 &a =   cudaGvtCtx->mesh.normals[normals.x];
          const float4 &b =   cudaGvtCtx->mesh.normals[normals.y];
          const float4 &c =   cudaGvtCtx->mesh.normals[normals.z];
          manualNormal = a * u + b * v + c * (1.0f - u - v);

          manualNormal =make_float4(
              (*(cudaGvtCtx->normi)) * make_float3(manualNormal.x,
            		  manualNormal.y,manualNormal.z));

          manualNormal=normalize(manualNormal);

#else
          int I = mesh->faces[triangle_id].get<0>();
          int J = mesh->faces[triangle_id].get<1>();
          int K = mesh->faces[triangle_id].get<2>();

          Vector4f a = mesh->vertices[I];
          Vector4f b = mesh->vertices[J];
          Vector4f c = mesh->vertices[K];
          Vector4f u = b - a;
          Vector4f v = c - a;
          Vector4f normal;
          normal.n[0] = u.n[1] * v.n[2] - u.n[2] * v.n[1];
          normal.n[1] = u.n[2] * v.n[0] - u.n[0] * v.n[2];
          normal.n[2] = u.n[0] * v.n[1] - u.n[1] * v.n[0];
          normal.n[3] = 0.0f;
          manualNormal = normal.normalize();
#endif
        }
        const float4 &normal = manualNormal;


        // reduce contribution of the color that the shadow rays get
        if (r.type == Ray::SECONDARY) {
          t = (t > 1) ? 1.f / t : t;
          r.w = r.w * t;
        }

       generateShadowRays(r, normal, cudaGvtCtx->
        		traceHits[tID].triangle_id, cudaGvtCtx);

        int ndepth = r.depth - 1;
        float p = 1.f - cudaRand();
        // replace current ray with generated secondary ray
        if (ndepth > 0 && r.w > p) {
          r.type = Ray::SECONDARY;
          const float multiplier =
              1.0f -
              16.0f *
                 FLT_EPSILON;

          const float t_secondary = multiplier * r.t;
          r.origin = r.origin + r.direction * t_secondary;
          r.origin.w=1.0f;

         float4 dir = normalize(cudaGvtCtx->mesh.mat->material.
                  		  CosWeightedRandomHemisphereDirection2(normal));

          r.setDirection(dir);

          r.w = r.w * (r.direction * normal);
          r.depth = ndepth;
          if (!cudaGvtCtx->validRayLeft)
        	  cudaGvtCtx->validRayLeft = true;

        } else {
        	cudaGvtCtx->valid[tID] = false;
        }
      } else {
        // ray is valid, but did not hit anything, so add to dispatch
    	  int a = atomicAdd((int *)&(cudaGvtCtx->dispatchCount), 1);
    	  cudaGvtCtx->dispatch[a] = r;

    	cudaGvtCtx->valid[tID] = false;

      }
    }

}

void shade(
		gvt::render::data::cuda_primitives::CudaGvtContext* cudaGvtCtx, const int startingDepth) {

	int N= cudaGvtCtx->rayCount;

	dim3 blockDIM = dim3(16, 16);
	dim3 gridDIM = dim3((N / (blockDIM.x * blockDIM.y)) + 1, 1);

	kernel<<<gridDIM,blockDIM , 0, cudaGvtCtx->stream >>>(cudaGvtCtx->toGPU(), startingDepth);
	  gpuErrchk(hipGetLastError());

	cudaGvtCtx->toHost();


}
