#include "hip/hip_runtime.h"


#include "cutil_math.h"
#include "Material.cuh"


using namespace gvt::render::data::cuda_primitives;





   __device__  cuda_vec BaseMaterial::CosWeightedRandomHemisphereDirection2(cuda_vec n) {

    float Xi1 = cudaRand();
    float Xi2 = cudaRand();

    float theta = acos(sqrt(1.0 - Xi1));
    float phi = 2.0 * 3.1415926535897932384626433832795 * Xi2;

    float xs = sinf(theta) * cosf(phi);
    float ys = cosf(theta);
    float zs = sinf(theta) * sinf(phi);

    float3 y = make_cuda_vec(n);
    float3 h = y;
    if (fabs(h.x) <= fabs(h.y) && fabs(h.x) <= fabs(h.z))
      h.x = 1.0;
    else if (fabs(h.y) <= fabs(h.x) && fabs(h.y) <= fabs(h.z))
      h.y = 1.0;
    else
      h.z = 1.0;

    float3 x = cross(h,y);//(h ^ y);
    float3 z = cross(x, y);

    cuda_vec direction = make_cuda_vec(x * xs + y * ys + z * zs);
    return normalize(direction);
  }



 /* Material::Material() {}s

  Material::Material(const Material &orig) {}

  Material::~Material() {}
*/
/*
  cuda_vec BaseMaterial::shade(const Ray &ray, const cuda_vec &sufaceNormal, const Light *lightSource) {
	  return make_cuda_vec(0.f);
  }
*/

  /*RayVector Material::ao(const Ray &ray, const cuda_vec &sufaceNormal, float samples) { return RayVector(); }

  RayVector Material::secondary(const Ray &ray, const cuda_vec &sufaceNormal, float samples) { return RayVector(); }

  Lambert::Lambert(const cuda_vec &kd) : Material(), kd(kd) {}

  Lambert::Lambert(const Lambert &orig) : Material(orig), kd(orig.kd) {}

  Lambert::~Lambert() {}
*/
   __device__ cuda_vec Lambert::shade( const Ray &ray, const cuda_vec &N, const Light *lightSource) {


	   cuda_vec hitPoint = ray.origin + ray.direction * ray.t;
	   cuda_vec L = normalize(lightSource->light.position - hitPoint);
    float NdotL = fmaxf(0.f, (N * L));
    cuda_vec lightSourceContrib = lightSource->contribution(hitPoint);

    cuda_vec diffuse = prod(lightSourceContrib, kd) * (NdotL * ray.w);

    return diffuse;
  }
/*
  RayVector Lambert::ao(const Ray &ray, const cuda_vec &sufaceNormal, float samples) { return RayVector(); }

  RayVector Lambert::secundary(const Ray &ray, const cuda_vec &sufaceNormal, float samples) { return RayVector(); }

  Phong::Phong(const cuda_vec &kd, const cuda_vec &ks, const float &alpha) : Material(), kd(kd), ks(ks), alpha(alpha) {}

  Phong::Phong(const Phong &orig) : Material(orig), kd(orig.kd), ks(orig.ks), alpha(orig.alpha) {}

  Phong::~Phong() {}

  */

   __device__ cuda_vec Phong::shade(const Ray &ray, const cuda_vec &N, const Light *lightSource) {


   cuda_vec hitPoint = ray.origin + (ray.direction * ray.t);
   cuda_vec L =normalize(lightSource->light.position - hitPoint);

    float NdotL =fmaxf(0.f, (N * L));
    cuda_vec R = ((N * 2.f) * NdotL) - L;
    cuda_vec invDir = make_cuda_vec(-ray.direction.x, -ray.direction.y, -ray.direction.z);
    float VdotR = fmaxf(0.f, (R * invDir));
    float power = VdotR * pow(VdotR, alpha);

    cuda_vec lightSourceContrib = lightSource->contribution(hitPoint); //  distance;

    Color finalColor = prod(lightSourceContrib , kd) * (NdotL * ray.w);
    finalColor += prod(lightSourceContrib , ks) * (power * ray.w);
    return finalColor;


    return finalColor;
  }

  /*

  RayVector Phong::ao(const Ray &ray, const cuda_vec &sufaceNormal, float samples) { return RayVector(); }

  RayVector Phong::secundary(const Ray &ray, const cuda_vec &sufaceNormal, float samples) { return RayVector(); }

  BlinnPhong::BlinnPhong(const cuda_vec &kd, const cuda_vec &ks, const float &alpha)
      : Material(), kd(kd), ks(ks), alpha(alpha) {}

  BlinnPhong::BlinnPhong(const BlinnPhong &orig) : Material(orig), kd(orig.kd), ks(orig.ks), alpha(orig.alpha) {}

  BlinnPhong::~BlinnPhong() {}
*/
   __device__ cuda_vec BlinnPhong::shade(const Ray &ray, const cuda_vec &N, const Light *lightSource) {
//    cuda_vec hitPoint = (cuda_vec)ray.origin + (ray.direction * ray.t);
//    cuda_vec L = (cuda_vec)lightSource->light.position - hitPoint;
//    L = normalize(L);
//    float NdotL = fmaxf(0.f, (N * L));
//
//    cuda_vec H = normalize((L - ray.direction));
//
//    float NdotH = (H * N);
//    float power = NdotH * std::pow(NdotH, alpha);
//
//    cuda_vec lightSourceContrib = lightSource->contribution(ray);
//
//    Color diffuse = prod((lightSourceContrib * NdotL), kd) * ray.w;
//    Color specular = prod((lightSourceContrib * power), ks) * ray.w;
//
//    Color finalColor = (diffuse + specular);
//    return finalColor;

	   cuda_vec hitPoint = ray.origin + (ray.direction * ray.t);
	   cuda_vec L = normalize(lightSource->light.position - hitPoint);
	   float NdotL = fmaxf(0.f, (N* L));

	   cuda_vec H = normalize(L - ray.direction);

	   float NdotH = fmaxf(0.f, (H * N));
	   float power = NdotH * pow(NdotH, alpha);

	   cuda_vec lightSourceContrib = lightSource->contribution(hitPoint);

	   Color diffuse = prod(lightSourceContrib , kd) * (NdotL * ray.w);
	   Color specular = prod(lightSourceContrib , ks) * (power * ray.w);

	   Color finalColor = (diffuse + specular);
	   return finalColor;
  }
/*
  RayVector BlinnPhong::ao(const Ray &ray, const cuda_vec &sufaceNormal, float samples) { return RayVector(); }

  RayVector BlinnPhong::secundary(const Ray &ray, const cuda_vec &sufaceNormal, float samples) { return RayVector(); }*/
